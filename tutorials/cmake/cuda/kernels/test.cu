#include "hip/hip_runtime.h"
/****************************************************************
 **** taken from: https://stackoverflow.com/questions/62139760/using-cmake-for-compiling-c-with-cuda-code
 ****************************************************************/

#include "test.cuh"
__global__ void test_print(){
  printf("Hello World!\n");
}

void wrap_test_print() {
  test_print<<<1, 1>>>();
  return;
}